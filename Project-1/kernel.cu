#include <hip/hip_runtime.h>


#include <stdio.h>

__global__ void hello_cuda()
{
	printf("threadIdx.x: %d, threadIdx.y: %d, threadIdx.z: %d, blockIdx.x: %d, blockIdx.y: %d, blockIdx.z: %d, gridDim.x: %d, gridDim.y: %d, gridDim.z: %d", threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z, gridDim.x, gridDim.y, gridDim.z);
}

int main()
{
	int nx, ny, nz;
	nx = 8;
	ny = 8;
	nz = 8;

	dim3 block(4, 4, 4);
	dim3 grid(nx / block.x, ny / block.y, nz / block.z);

	hello_cuda<<<grid, block>>> ();
	hipDeviceSynchronize();

	hipDeviceReset();

	return 0;
}