#include <stdlib.h>
#include <stdio.h>

_global_ void print_from_gpu(void) {
    printf("Hello World! from thread [%d,%d] \
    From device\n", threadIdx.x, BlockIdx.x);
}

int main(void) {
    printf("Hello World from host!\n");
    print_from_gpu<<<1,1>>>();
    hipDeviceSynchronize();
return 0;
}