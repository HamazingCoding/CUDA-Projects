#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>

__global__ void unique_grid_calculation_2d(int * data)
{
	int tid = threadIdx.x;
	int offset = blockIdx.x * blockDim.x;
	int gid = tid + offset;
	printf("blockIdx.x: %d, blockIdx.y: %d, threadIdx.x: %d, gid : %d - data : %d \n", 
		blockIdx.x, blockIdx.y, tid, gid, data[gid]);
}

int main()
{
	int array_size = 16;
	int array_byte_size = sizeof(int) * array_size;
	int h_data[] = { 23,9,4,53,65,12,1,33,22,43,56,4,76,81,94,32 };

	int* d_data;
	hipMalloc((void**)&d_data, array_byte_size);
	hipMemcpy(d_data, h_data, array_byte_size, hipMemcpyHostToDevice);

	dim3 block(4);
	dim3 grid(4);

	unique_grid_calculation_2d << <grid, block>> > (d_data);
	hipDeviceSynchronize();

	hipDeviceReset();

	return 0;
}